#include "cuda_defines.h"
#include "cuda_stuff.h"

void CUDA_Initialize(const int sx, const int sy, const int sz, const int bord,
                     float dx, float dy, float dz, float dt,
                     float *restrict ch1dxx, float *restrict ch1dyy, float *restrict ch1dzz,
                     float *restrict ch1dxy, float *restrict ch1dyz, float *restrict ch1dxz,
                     float *restrict v2px, float *restrict v2pz, float *restrict v2sz, float *restrict v2pn,
                     float *restrict vpz, float *restrict vsv, float *restrict epsilon, float *restrict delta,
                     float *restrict phi, float *restrict theta,
                     float *restrict pp, float *restrict pc, float *restrict qp, float *restrict qc)
{

   int deviceCount;
   CUDA_CALL(hipGetDeviceCount(&deviceCount));
   const int device = 0;
   hipDeviceProp_t deviceProp;
   CUDA_CALL(hipGetDeviceProperties(&deviceProp, device));
   printf("CUDA source using device(%d) %s with compute capability %d.%d.\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
   CUDA_CALL(hipSetDevice(device));

   // Check sx,sy values
   if (sx % BSIZE_X != 0)
   {
      printf("sx(%d) must be multiple of BSIZE_X(%d)\n", sx, (int)BSIZE_X);
      exit(1);
   }
   if (sy % BSIZE_Y != 0)
   {
      printf("sy(%d) must be multiple of BSIZE_Y(%d)\n", sy, (int)BSIZE_Y);
      exit(1);
   }

   int sxsy = sx * sy; // one plan
   const size_t sxsysz = sxsy * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sxsy * sizeof(float); // 2 extra plans for wave fields

   // arthur -- Se for realizar a cópia assíncrona com prefetch, é aqui o lugar.

   hipMemPrefetchAsync(ch1dxx, msize_vol, device);
   hipMemPrefetchAsync(ch1dyy, msize_vol, device);
   hipMemPrefetchAsync(ch1dzz, msize_vol, device);
   hipMemPrefetchAsync(ch1dxy, msize_vol, device);
   hipMemPrefetchAsync(ch1dyz, msize_vol, device);
   hipMemPrefetchAsync(ch1dxz, msize_vol, device);
   hipMemPrefetchAsync(v2px, msize_vol, device);
   hipMemPrefetchAsync(v2pz, msize_vol, device);
   hipMemPrefetchAsync(v2sz, msize_vol, device);
   hipMemPrefetchAsync(v2pn, msize_vol, device);

   hipMemPrefetchAsync(pp, msize_vol_extra, device);
   hipMemPrefetchAsync(pc, msize_vol_extra, device);
   hipMemPrefetchAsync(qp, msize_vol_extra, device);
   hipMemPrefetchAsync(qc, msize_vol_extra, device);

   CUDA_CALL(hipGetLastError());
   // CUDA_CALL(hipDeviceSynchronize());

   pp += sxsy;
   pc += sxsy;
   qp += sxsy;
   qc += sxsy;

   printf("GPU memory usage = %ld MiB\n", 15 * msize_vol / 1024 / 1024);

   size_t freeMem, totalMem;
   CUDA_CALL(hipMemGetInfo(&freeMem, &totalMem));
   printf("GPU memory usage: %lu MiB (used) / %lu MiB (total)\n", (totalMem - freeMem) / (1024 * 1024), totalMem / (1024 * 1024));
}

// ARTHUR - Ajustar função para receber os parametros do CUDA_Finalize.
void CUDA_Finalize(const int sx, const int sy, const int sz, const int bord,
                   float dx, float dy, float dz, float dt,
                   float *restrict ch1dxx, float *restrict ch1dyy, float *restrict ch1dzz,
                   float *restrict ch1dxy, float *restrict ch1dyz, float *restrict ch1dxz,
                   float *restrict v2px, float *restrict v2pz, float *restrict v2sz, float *restrict v2pn,
                   float *restrict vpz, float *restrict vsv, float *restrict epsilon, float *restrict delta,
                   float *restrict phi, float *restrict theta,
                   float *restrict pp, float *restrict pc, float *restrict qp, float *restrict qc)
{
   int sxsy = sx * sy; // one plan
   pp -= sxsy;
   pc -= sxsy;
   qp -= sxsy;
   qc -= sxsy;

   CUDA_CALL(hipFree(vpz));
   CUDA_CALL(hipFree(vsv));
   CUDA_CALL(hipFree(epsilon));
   CUDA_CALL(hipFree(delta));
   CUDA_CALL(hipFree(phi));
   CUDA_CALL(hipFree(theta));
   CUDA_CALL(hipFree(ch1dxx));
   CUDA_CALL(hipFree(ch1dyy));
   CUDA_CALL(hipFree(ch1dzz));
   CUDA_CALL(hipFree(ch1dxy));
   CUDA_CALL(hipFree(ch1dyz));
   CUDA_CALL(hipFree(ch1dxz));
   CUDA_CALL(hipFree(v2px));
   CUDA_CALL(hipFree(v2pz));
   CUDA_CALL(hipFree(v2sz));
   CUDA_CALL(hipFree(v2pn));
   // CUDA_CALL(hipFree(pp));
   // CUDA_CALL(hipFree(pc));
   // CUDA_CALL(hipFree(qp));
   // CUDA_CALL(hipFree(qc));

   printf("CUDA_Finalize: SUCCESS\n");
}

void CUDA_Update_pointers(const int sx, const int sy, const int sz, float *pc)
{
   // arthur: se usar memoria unificada, não precisa desta cópia.
   // extern float* dev_pc;
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   // if (pc) CUDA_CALL(hipMemcpy(pc, dev_pc, msize_vol, hipMemcpyDeviceToHost));
   CUDA_CALL(hipMemPrefetchAsync(pc, msize_vol, hipCpuDeviceId));
}

void CUDA_Allocate_Model_Variables(float **restrict ch1dxx, float **restrict ch1dyy, float **restrict ch1dzz, float **restrict ch1dxy,
                                   float **restrict ch1dyz, float **restrict ch1dxz, float **restrict v2px, float **restrict v2pz, float **restrict v2sz,
                                   float **restrict v2pn, int sx, int sy, int sz)
{
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   CUDA_CALL(hipMallocManaged(ch1dxx, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dyy, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dzz, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dxy, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dyz, msize_vol));
   CUDA_CALL(hipMallocManaged(ch1dxz, msize_vol));
   CUDA_CALL(hipMallocManaged(v2px, msize_vol));
   CUDA_CALL(hipMallocManaged(v2pz, msize_vol));
   CUDA_CALL(hipMallocManaged(v2sz, msize_vol));
   CUDA_CALL(hipMallocManaged(v2pn, msize_vol));
}

void CUDA_Allocate_main(float **restrict vpz, float **restrict vsv, float **restrict epsilon, float **restrict delta,
                        float **restrict phi, float **restrict theta, float **restrict pp, float **restrict pc, float **restrict qp,
                        float **restrict qc, int sx, int sy, int sz)
{
   int sxsy = sx * sy;
   const size_t sxsysz = ((size_t)sx * sy) * sz;
   const size_t msize_vol = sxsysz * sizeof(float);
   const size_t msize_vol_extra = msize_vol + 2 * sx * sy * sizeof(float); // 2 extra plans for wave fields
   CUDA_CALL(hipMallocManaged(vpz, msize_vol));
   CUDA_CALL(hipMallocManaged(vsv, msize_vol));
   CUDA_CALL(hipMallocManaged(epsilon, msize_vol));
   CUDA_CALL(hipMallocManaged(delta, msize_vol));
   CUDA_CALL(hipMallocManaged(phi, msize_vol));
   CUDA_CALL(hipMallocManaged(theta, msize_vol));

   CUDA_CALL(hipMallocManaged(pp, msize_vol_extra));
   CUDA_CALL(hipMallocManaged(pc, msize_vol_extra));
   CUDA_CALL(hipMallocManaged(qp, msize_vol_extra));
   CUDA_CALL(hipMallocManaged(qc, msize_vol_extra));
   // ARTHUR - Ver se esta operação fica na CPU ou mover para a GPU.
   memset(*pp, 0, msize_vol_extra);
   memset(*pc, 0, msize_vol_extra);
   memset(*qp, 0, msize_vol_extra);
   memset(*qc, 0, msize_vol_extra);
   // pp+=sxsy;
   // pc+=sxsy;
   // qp+=sxsy;
   // qc+=sxsy;
}
